#include "hip/hip_runtime.h"
// Code by Michael Katz. Based on code by Travis Robson, Neil Cornish, Tyson Littenberg, Stas Babak

// imports
#include "stdio.h"

#include "gbgpu_utils.hh"
#include "global.h"
#include "LISA.h"
#include "cuda_complex.hpp"

#ifdef __HIPCC__
#include "hip/hip_complex.h"
#include "hipblas.h"
#else
#include <gsl/gsl_errno.h>
#include <gsl/gsl_sf_bessel.h>
#include <gsl/gsl_cblas.h>
#endif

#define NUM_THREADS 256

// Add functionality for proper summation in the kernel
#ifdef __HIPCC__
CUDA_DEVICE
double atomicAddDouble(double *address, double val)
{
    unsigned long long *address_as_ull =
        (unsigned long long *)address;
    unsigned long long old = *address_as_ull, assumed;

    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

// Add functionality for proper summation in the kernel
CUDA_DEVICE
void atomicAddComplex(cmplx *a, cmplx b)
{
    // transform the addresses of real and imag. parts to double pointers
    double *x = (double *)a;
    double *y = x + 1;
    // use atomicAdd for double variables

#ifdef __HIPCC__
    atomicAddDouble(x, b.real());
    atomicAddDouble(y, b.imag());
#else
    *x += b.real();
    *y += b.imag();
#endif
}

// calculate batched log likelihood
CUDA_KERNEL
void fill_global(cmplx *A_glob, cmplx *E_glob, cmplx *A_template, cmplx *E_template, int *start_ind_all, int M, int num_bin, int *group_index, int data_length)
{
    // prepare loop based on CPU/GPU
    int start, end, increment;
#ifdef __HIPCC__

    start = blockIdx.x * blockDim.x + threadIdx.x;
    end = num_bin;
    increment = blockDim.x * gridDim.x;

#else

    start = 0;
    end = num_bin;
    increment = 1;

#endif
    for (int bin_i = start;
         bin_i < end;
         bin_i += increment)
    {

        // get start index in frequency array
        int start_ind = start_ind_all[bin_i];
        int group_i = group_index[bin_i];

        for (int i = 0;
             i < M;
             i += 1)
        {
            int j = start_ind + i;

            if ((j >= data_length) || (j < 0))
            {
                continue;
            }
            cmplx temp_A = A_template[i * num_bin + bin_i];
            cmplx temp_E = E_template[i * num_bin + bin_i];

            int ind_out = group_i * data_length + j;
            atomicAddComplex(&A_glob[ind_out], temp_A);
            atomicAddComplex(&E_glob[ind_out], temp_E);
            // printf("CHECK: %d %e %e %d %d %d %d %d %d\n", bin_i, A_template[i * num_bin + bin_i], temp_A, group_i, data_length, j, num_groups, per_group, i);
        }
    }
}

// wrapper for log likelihood
void fill_global_wrap(cmplx *A_glob, cmplx *E_glob, cmplx *A_template, cmplx *E_template, int *start_ind_all, int M, int num_bin, int *group_index, int data_length)
{
// GPU / CPU difference
#ifdef __HIPCC__

    int num_blocks = std::ceil((num_bin + NUM_THREADS - 1) / NUM_THREADS);

    if (num_blocks == 0) return;

    fill_global<<<num_blocks, NUM_THREADS>>>(
        A_glob, E_glob, A_template, E_template, start_ind_all, M, num_bin, group_index, data_length);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

#else

    fill_global(
        A_glob, E_glob, A_template, E_template, start_ind_all, M, num_bin, group_index, data_length);

#endif
}

// calculate batched log likelihood
CUDA_KERNEL
void get_ll(cmplx *d_h, cmplx *h_h, cmplx *A_template, cmplx *E_template, cmplx *A_data, cmplx *E_data, double *A_psd, double *E_psd, double df, int *start_ind_all, int M, int num_bin, int *data_index, int *noise_index, int data_length)
{
    // prepare loop based on CPU/GPU
    int start, end, increment;
#ifdef __HIPCC__

    start = blockIdx.x * blockDim.x + threadIdx.x;
    end = num_bin;
    increment = blockDim.x * gridDim.x;

#else

    start = 0;
    end = num_bin;
    increment = 1;

#endif
    for (int bin_i = start;
         bin_i < end;
         bin_i += increment)
    {

        // get start index in frequency array
        int start_ind = start_ind_all[bin_i];
        int data_index_bin_i = data_index[bin_i];
        int noise_index_bin_i = noise_index[bin_i];

        // initialize likelihood
        cmplx h_h_temp(0.0, 0.0);
        cmplx d_h_temp(0.0, 0.0);
        for (int i = 0;
             i < M;
             i += 1)
        {
            int j = start_ind + i;

            double A_noise = A_psd[noise_index_bin_i * data_length + j];
            double E_noise = E_psd[noise_index_bin_i * data_length + j];

            // calculate h term
            cmplx h_A = A_template[i * num_bin + bin_i];
            cmplx h_E = E_template[i * num_bin + bin_i];

            cmplx d_A = A_data[data_index_bin_i * data_length + j];
            cmplx d_E = E_data[data_index_bin_i * data_length + j];

            // get <d|h> term
            d_h_temp += gcmplx::conj(d_A) * h_A / A_noise;
            d_h_temp += gcmplx::conj(d_E) * h_E / E_noise;

            // <h|h>
            h_h_temp += gcmplx::conj(h_A) * h_A / A_noise;
            h_h_temp += gcmplx::conj(h_E) * h_E / E_noise;
        }

        // read out
        d_h[bin_i] = 4. * df * d_h_temp;
        h_h[bin_i] = 4. * df * h_h_temp;
    }
}

// wrapper for log likelihood
void get_ll_wrap(cmplx *d_h, cmplx *h_h,
                 cmplx *A_template, cmplx *E_template,
                 cmplx *A_data, cmplx *E_data,
                 double *A_psd, double *E_psd, double df,
                 int *start_ind, int M, int num_bin, int *data_index, int *noise_index, int data_length)
{
// GPU / CPU difference
#ifdef __HIPCC__

    int num_blocks = std::ceil((num_bin + NUM_THREADS - 1) / NUM_THREADS);

    get_ll<<<num_blocks, NUM_THREADS>>>(
        d_h, h_h, A_template, E_template, A_data, E_data, A_psd, E_psd, df, start_ind, M, num_bin, data_index, noise_index, data_length);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

#else

    get_ll(
        d_h, h_h, A_template, E_template, A_data, E_data, A_psd, E_psd, df, start_ind, M, num_bin, data_index, noise_index, data_length);

#endif
}

#ifdef __HIPCC__
void direct_like(cmplx *d_h, cmplx *h_h,
                 cmplx *A_template, cmplx *E_template,
                 cmplx *A_data, cmplx *E_data,
                 int data_length, int start_freq_ind, int nwalkers)
{

    hipStream_t streams[nwalkers];
    hipblasHandle_t handle;

    hipDoubleComplex result_d_h[nwalkers];
    hipDoubleComplex result_h_h[nwalkers];

    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("CUBLAS initialization failed\n");
        exit(0);
    }

    for (int walker_i = 0; walker_i < nwalkers; walker_i += 1)
    {

        hipStreamCreate(&streams[walker_i]);

        hipblasSetStream(handle, streams[walker_i]);
        stat = hipblasZdotc(handle, data_length,
                           (hipDoubleComplex *)&A_data[start_freq_ind], 1,
                           (hipDoubleComplex *)&A_template[walker_i * data_length], 1,
                           &result_d_h[walker_i]);
        hipStreamSynchronize(streams[walker_i]);
        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            exit(0);
        }

        cmplx *temp_results_d_h = (cmplx *)&result_d_h[walker_i];
        d_h[walker_i] += 4.0 * (*temp_results_d_h);

        hipblasSetStream(handle, streams[walker_i]);
        stat = hipblasZdotc(handle, data_length,
                           (hipDoubleComplex *)&A_template[walker_i * data_length], 1,
                           (hipDoubleComplex *)&A_template[walker_i * data_length], 1,
                           &result_h_h[walker_i]);
        hipStreamSynchronize(streams[walker_i]);
        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            exit(0);
        }

        cmplx *temp_results_h_h = (cmplx *)&result_h_h[walker_i];
        h_h[walker_i] += 4.0 * (*temp_results_h_h);

        hipblasSetStream(handle, streams[walker_i]);
        stat = hipblasZdotc(handle, data_length,
                           (hipDoubleComplex *)&E_data[start_freq_ind], 1,
                           (hipDoubleComplex *)&E_template[walker_i * data_length], 1,
                           &result_d_h[walker_i]);
        hipStreamSynchronize(streams[walker_i]);
        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            exit(0);
        }

        temp_results_d_h = (cmplx *)&result_d_h[walker_i];
        d_h[walker_i] += 4.0 * (*temp_results_d_h);

        hipblasSetStream(handle, streams[walker_i]);
        stat = hipblasZdotc(handle, data_length,
                           (hipDoubleComplex *)&E_template[walker_i * data_length], 1,
                           (hipDoubleComplex *)&E_template[walker_i * data_length], 1,
                           &result_h_h[walker_i]);
        hipStreamSynchronize(streams[walker_i]);
        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            exit(0);
        }

        temp_results_h_h = (cmplx *)&result_h_h[walker_i];
        h_h[walker_i] += 4.0 * (*temp_results_h_h);
    }

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    for (int walker_i = 0; walker_i < nwalkers; walker_i += 1)
    {
        // destroy the streams
        hipStreamDestroy(streams[walker_i]);
    }
    hipblasDestroy(handle);
}

#else
void direct_like(cmplx *d_h, cmplx *h_h,
                 cmplx *A_template, cmplx *E_template,
                 cmplx *A_data, cmplx *E_data,
                 int data_length, int start_freq_ind, int nwalkers)
{

    cmplx result_d_h[nwalkers];
    cmplx result_h_h[nwalkers];

    for (int walker_i = 0; walker_i < nwalkers; walker_i += 1)
    {

        cblas_zdotc_sub(data_length,
                        (void *)&A_data[start_freq_ind], 1,
                        (void *)&A_template[walker_i * data_length], 1,
                        (void *)&result_d_h[walker_i]);

        d_h[walker_i] += 4.0 * result_d_h[walker_i];

        cblas_zdotc_sub(data_length,
                        (void *)&A_template[walker_i * data_length], 1,
                        (void *)&A_template[walker_i * data_length], 1,
                        (void *)&result_h_h[walker_i]);

        h_h[walker_i] += 4.0 * result_h_h[walker_i];

        cblas_zdotc_sub(data_length,
                        (void *)&E_data[start_freq_ind], 1,
                        (void *)&E_template[walker_i * data_length], 1,
                        (void *)&result_d_h[walker_i]);

        d_h[walker_i] += 4.0 * result_d_h[walker_i];

        cblas_zdotc_sub(data_length,
                        (void *)&E_template[walker_i * data_length], 1,
                        (void *)&E_template[walker_i * data_length], 1,
                        (void *)&result_h_h[walker_i]);

        h_h[walker_i] += 4.0 * result_h_h[walker_i];
    }
}
#endif


// calculate batched log likelihood
CUDA_KERNEL
void swap_ll_diff(cmplx* d_h_remove, cmplx* d_h_add, cmplx* add_remove, cmplx* remove_remove, cmplx* add_add, cmplx* A_remove, cmplx* E_remove, int* start_ind_all_remove, cmplx* A_add, cmplx* E_add, int* start_ind_all_add, cmplx* A_data, cmplx* E_data, double* A_psd, double* E_psd, double df, int M, int num_bin, int* data_index, int* noise_index, int data_length)
{
    // prepare loop based on CPU/GPU
    int start, end, increment;
    #ifdef __HIPCC__

    start = blockIdx.x * blockDim.x + threadIdx.x;
    end = num_bin;
    increment = blockDim.x * gridDim.x;

    #else

    start = 0;
    end = num_bin;
    increment = 1;

    #pragma omp parallel for
    #endif
	for (int bin_i = start;
			 bin_i < end;
			 bin_i += increment)
    {
        
        // get start index in frequency array
        int start_ind_remove = start_ind_all_remove[bin_i];
        int start_ind_add = start_ind_all_add[bin_i];
        int data_index_bin_i = data_index[bin_i];
        int noise_index_bin_i = noise_index[bin_i];

        // initialize likelihood
        cmplx d_h_remove_temp(0.0, 0.0);
        cmplx d_h_add_temp(0.0, 0.0);
        cmplx add_remove_temp(0.0, 0.0);
        cmplx remove_remove_temp(0.0, 0.0);
        cmplx add_add_temp(0.0, 0.0);

        int lower_start_ind, upper_start_ind, lower_end_ind, upper_end_ind;
        bool is_add_lower;
        if (start_ind_remove <= start_ind_add)
        {
            lower_start_ind = start_ind_remove;
            upper_end_ind = start_ind_add + M;

            upper_start_ind = start_ind_add;
            lower_end_ind = start_ind_remove +M;

            is_add_lower = false;
        }
        else
        {
            lower_start_ind = start_ind_add;
            upper_end_ind = start_ind_remove + M;

            upper_start_ind = start_ind_remove;
            lower_end_ind = start_ind_add + M;

            is_add_lower = true;
        }
        int total_i_vals = upper_end_ind - lower_start_ind;

        double A_noise, E_noise;
        cmplx d_A, d_E;

        cmplx h_A, h_E, h_A_add, h_E_add, h_A_remove, h_E_remove;
        int real_ind, real_ind_add, real_ind_remove; 
        
        if (total_i_vals < 2 * M)
        {
            for (int i = 0;
                    i < total_i_vals;
                    i += 1)
            {
                
                int j = lower_start_ind + i;

                
                A_noise = A_psd[noise_index_bin_i * data_length + j];
                E_noise = E_psd[noise_index_bin_i * data_length + j];

                d_A = A_data[data_index_bin_i * data_length + j];
                d_E = E_data[data_index_bin_i * data_length + j];

                
                
                if (j < upper_start_ind)
                {
                    real_ind = i;
                    if (is_add_lower)
                    {
                        
                        h_A = A_add[real_ind *num_bin + bin_i];
                        h_E = E_add[real_ind *num_bin + bin_i];

                        // get <d|h> term
                        d_h_add_temp += gcmplx::conj(d_A) * h_A / A_noise;
                        d_h_add_temp += gcmplx::conj(d_E) * h_E / E_noise;

                        // <h|h>
                        add_add_temp += gcmplx::conj(h_A) * h_A / A_noise;
                        add_add_temp += gcmplx::conj(h_E) * h_E / E_noise;
                    }
                    else
                    {
                        h_A = A_remove[real_ind *num_bin + bin_i];
                        h_E = E_remove[real_ind *num_bin + bin_i];

                        // get <d|h> term
                        d_h_remove_temp += gcmplx::conj(d_A) * h_A / A_noise;
                        d_h_remove_temp += gcmplx::conj(d_E) * h_E / E_noise;

                        // <h|h>
                        remove_remove_temp += gcmplx::conj(h_A) * h_A / A_noise;
                        remove_remove_temp += gcmplx::conj(h_E) * h_E / E_noise;

                        //if ((bin_i == 0)) printf("%d %d %d \n", i, j, upper_start_ind);
                    }
                }
                else if (j >= lower_end_ind)
                {
                    real_ind = j - upper_start_ind;
                    if (!is_add_lower)
                    {

                        h_A_add = A_add[real_ind *num_bin + bin_i];
                        h_E_add = E_add[real_ind *num_bin + bin_i];

                        // get <d|h> term
                        d_h_add_temp += gcmplx::conj(d_A) * h_A_add / A_noise;
                        d_h_add_temp += gcmplx::conj(d_E) * h_E_add / E_noise;

                        // <h|h>
                        add_add_temp += gcmplx::conj(h_A_add) * h_A_add / A_noise;
                        add_add_temp += gcmplx::conj(h_E_add) * h_E_add / E_noise;
                    }
                    else
                    {
                        h_A_remove = A_remove[real_ind *num_bin + bin_i];
                        h_E_remove = E_remove[real_ind *num_bin + bin_i];

                        // get <d|h> term
                        d_h_remove_temp += gcmplx::conj(d_A) * h_A_remove / A_noise;
                        d_h_remove_temp += gcmplx::conj(d_E) * h_E_remove / E_noise;

                        // <h|h>
                        remove_remove_temp += gcmplx::conj(h_A_remove) * h_A_remove / A_noise;
                        remove_remove_temp += gcmplx::conj(h_E_remove) * h_E_remove / E_noise;
                    }
                }
                else // this is where the signals overlap
                {
                    if (is_add_lower)
                    {
                        real_ind_add = i;
                    }
                    else
                    {
                        real_ind_add = j - upper_start_ind;
                    }

                    h_A_add = A_add[real_ind_add *num_bin + bin_i];
                    h_E_add = E_add[real_ind_add *num_bin + bin_i];

                    // get <d|h> term
                    d_h_add_temp += gcmplx::conj(d_A) * h_A_add / A_noise;
                    d_h_add_temp += gcmplx::conj(d_E) * h_E_add / E_noise;

                    // <h|h>
                    add_add_temp += gcmplx::conj(h_A_add) * h_A_add / A_noise;
                    add_add_temp += gcmplx::conj(h_E_add) * h_E_add / E_noise;

                    if (!is_add_lower)
                    {
                        real_ind_remove = i;
                    }
                    else
                    {
                        real_ind_remove = j - upper_start_ind;
                    }
                    
                    h_A_remove = A_remove[real_ind_remove *num_bin + bin_i];
                    h_E_remove = E_remove[real_ind_remove *num_bin + bin_i];

                    // get <d|h> term
                    d_h_remove_temp += gcmplx::conj(d_A) * h_A_remove / A_noise;
                    d_h_remove_temp += gcmplx::conj(d_E) * h_E_remove / E_noise;

                    // <h|h>
                    remove_remove_temp += gcmplx::conj(h_A_remove) * h_A_remove / A_noise;
                    remove_remove_temp += gcmplx::conj(h_E_remove) * h_E_remove / E_noise;

                    add_remove_temp += gcmplx::conj(h_A_add) * h_A_remove / A_noise;
                    add_remove_temp += gcmplx::conj(h_E_add) * h_E_remove / E_noise;
                }
            }
        }
        else
        {
            for (int i = 0;
                    i < M;
                    i += 1)
            {
                
                int j = start_ind_remove + i;

                
                A_noise = A_psd[noise_index_bin_i * data_length + j];
                E_noise = E_psd[noise_index_bin_i * data_length + j];

                d_A = A_data[data_index_bin_i * data_length + j];
                d_E = E_data[data_index_bin_i * data_length + j];

                //if ((bin_i == num_bin - 1))printf("CHECK remove: %d %e %e  \n", i, A_noise, d_A.real());
                // calculate h term
                h_A = A_remove[i * num_bin + bin_i];
                h_E = E_remove[i * num_bin + bin_i];

                // get <d|h> term
                d_h_remove_temp += gcmplx::conj(d_A) * h_A / A_noise;
                d_h_remove_temp += gcmplx::conj(d_E) * h_E / E_noise;

                // <h|h>
                remove_remove_temp += gcmplx::conj(h_A) * h_A / A_noise;
                remove_remove_temp += gcmplx::conj(h_E) * h_E / E_noise;
                
            }

            for (int i = 0;
                    i < M;
                    i += 1)
            {
                
                int j = start_ind_add + i;

                
                A_noise = A_psd[noise_index_bin_i * data_length + j];
                E_noise = E_psd[noise_index_bin_i * data_length + j];

                d_A = A_data[data_index_bin_i * data_length + j];
                d_E = E_data[data_index_bin_i * data_length + j];

                //if ((bin_i == num_bin - 1))printf("CHECK add: %d %e %e  \n", i, A_noise, d_A.real());
                // calculate h term
                h_A = A_add[i * num_bin + bin_i];
                h_E = E_add[i * num_bin + bin_i];

                // get <d|h> term
                d_h_add_temp += gcmplx::conj(d_A) * h_A / A_noise;
                d_h_add_temp += gcmplx::conj(d_E) * h_E / E_noise;

                // <h|h>
                add_add_temp += gcmplx::conj(h_A) * h_A / A_noise;
                add_add_temp += gcmplx::conj(h_E) * h_E / E_noise;
                
            }
        }
        
        // read out
        d_h_remove[bin_i] =  4. * df * d_h_remove_temp;
        d_h_add[bin_i] =  4. * df * d_h_add_temp;
        add_add[bin_i] =  4. * df * add_add_temp;
        add_remove[bin_i] =  4. * df * add_remove_temp;
        remove_remove[bin_i] =  4. * df * remove_remove_temp;
        
    }
}


// wrapper for log likelihood
void swap_ll_diff_wrap(cmplx* d_h_remove, cmplx* d_h_add, cmplx* add_remove, cmplx* remove_remove, cmplx* add_add, cmplx* A_remove, cmplx* E_remove, int* start_ind_all_remove, cmplx* A_add, cmplx* E_add, int* start_ind_all_add, cmplx* A_data, cmplx* E_data, double* A_psd, double* E_psd, double df, int M, int num_bin, int* data_index, int* noise_index, int data_length)
{
    // GPU / CPU difference
    #ifdef __HIPCC__

    int num_blocks = std::ceil((num_bin + NUM_THREADS -1)/NUM_THREADS);

    swap_ll_diff<<<num_blocks, NUM_THREADS>>>(
       d_h_remove, d_h_add, add_remove, remove_remove, add_add, A_remove, E_remove, start_ind_all_remove, A_add, E_add, start_ind_all_add, A_data, E_data, A_psd, E_psd, df, M, num_bin, data_index, noise_index, data_length
    );
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    #else

    swap_ll_diff(
       d_h_remove, d_h_add, add_remove, remove_remove, add_add, A_remove, E_remove, start_ind_all_remove, A_add, E_add, start_ind_all_add, A_data, E_data, A_psd, E_psd, df, M, num_bin, data_index, noise_index, data_length
    );

    #endif
}
